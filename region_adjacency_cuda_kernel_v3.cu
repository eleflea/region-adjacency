#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__constant__ int NEIGHBOR_TABLE[12];

template <typename scalar_t, int neighbor_count>
__global__ void region_adjacency_cuda_forward_kernel(const scalar_t *__restrict__ labelled_imgs,
                                                     uint8_t *__restrict__ output_adj, const size_t label_size,
                                                     const int height, const int width) {
  const auto x = threadIdx.x + blockDim.x * blockIdx.x;
  const auto y = threadIdx.y + blockDim.y * blockIdx.y;
  const auto index = (blockIdx.z * height + y) * width + x;
  if (x < width && y < height) {
    // calculate neighbor index (other_i) based on sub_i
    // -------------
    // | 2 | 1 | 3 |
    // -------------
    // | 0 |   | X |
    // -------------
    // | X | X | X |
    // -------------
    const auto v = labelled_imgs[index];
    for (size_t i = 0; i < neighbor_count; ++i) {
      const auto neighbor = NEIGHBOR_TABLE + i * 3;
      auto other_index = index + (x != neighbor[0] && y != neighbor[1]) * neighbor[2];

      const auto nv = labelled_imgs[other_index];
      const auto pbase = output_adj + blockIdx.z * label_size * label_size;
      const auto pa = pbase + v * label_size + nv;
      const auto pb = pbase + nv * label_size + v;
      if (nv != v && *pa == 0) {
        *pa = 1;
        *pb = 1;
      }
    }
  }
}

torch::Tensor region_adjacency_cuda_forward(const torch::Tensor labelled_imgs, const int num_labels,
                                            const int connectivity) {
  const auto batch_size = labelled_imgs.size(0);
  const int height = labelled_imgs.size(1);
  const int width = labelled_imgs.size(2);
  const size_t label_size = num_labels == 0 ? labelled_imgs.max().item<int64_t>() + 1 : num_labels;
  auto output_adj = torch::zeros({batch_size, num_labels, num_labels}, labelled_imgs.options().dtype(torch::kUInt8));

  constexpr auto tile_size = 32;
  const dim3 threads(tile_size, tile_size);
  const dim3 blocks((width + tile_size - 1) / tile_size, (height + tile_size - 1) / tile_size, batch_size);

  const int neighbor_table[] = {0, -1, -1, -1, 0, -width, 0, 0, -width - 1, width - 1, 0, -width + 1};
  hipMemcpyToSymbol(HIP_SYMBOL(NEIGHBOR_TABLE), neighbor_table, 12 * sizeof(int));

  if (connectivity == 1) {
    AT_DISPATCH_INTEGRAL_TYPES(labelled_imgs.type(), "region_adjacency_forward_cuda", ([&] {
                                 region_adjacency_cuda_forward_kernel<scalar_t, 2>
                                   <<<blocks, threads>>>(labelled_imgs.data<scalar_t>(), output_adj.data<uint8_t>(),
                                                         label_size, height, width);
                               }));
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(labelled_imgs.type(), "region_adjacency_forward_cuda", ([&] {
                                 region_adjacency_cuda_forward_kernel<scalar_t, 4>
                                   <<<blocks, threads>>>(labelled_imgs.data<scalar_t>(), output_adj.data<uint8_t>(),
                                                         label_size, height, width);
                               }));
  }

  output_adj = output_adj.to(labelled_imgs.dtype());

  return output_adj;
}
